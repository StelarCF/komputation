#include "hip/hip_runtime.h"
#include "zero/Zero.cuh"

__device__ float backwardSigmoid (float forward, float chain)
{

    return forward * (1.0f - forward) * chain;

}

extern "C"
__global__ void backwardSigmoidKernel (int batchSize, int numberEntriesPerInstance, int numberIterations, float *forward, float *chain, float *destination) {

    // What's the first entry index within the instance that this thread should operate on?
    int startIndexWithinInstance = blockIdx.y * (blockDim.x * numberIterations) + threadIdx.x * numberIterations;

    // Continue if this index is smaller than the dimension of the instance.
    if(startIndexWithinInstance < numberEntriesPerInstance) {

        // What's the first entry index within the batch that this thread should operate on?
        int startIndexWithinBatch = blockIdx.x * numberEntriesPerInstance + startIndexWithinInstance;

        // Is the instance greater than the current batch size?
        if(blockIdx.x >= batchSize) {

            setToZero(destination, startIndexWithinBatch, numberIterations);

        }
        else {

            for(int indexEntry = startIndexWithinBatch; indexEntry < startIndexWithinBatch + numberIterations; indexEntry++) {

                destination[indexEntry] = backwardSigmoid(forward[indexEntry], chain[indexEntry]);

            }

        }

    }

}