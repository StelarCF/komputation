
#include <hip/hip_runtime.h>
extern "C"
__global__ void stochasticGradientDescentKernel (int length, float *parameter, float scalingFactor, float learningRate, float *gradient)
{

    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index < length) {

        parameter[index] = parameter[index] - scalingFactor * learningRate * gradient[index];
        gradient[index] = 0.0;

    }

    __syncthreads();

}