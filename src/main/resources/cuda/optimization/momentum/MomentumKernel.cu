
#include <hip/hip_runtime.h>
extern "C"
__global__ void momentumKernel (
    int numberIterations,
    float learningRate,
    float momentum,
    float* history,
    int* parameterIndices,
    int parameterSize,
    float* parameters,
    float scalingFactor,
    float* gradient)
{

    int startEntry = (blockIdx.y * blockDim.x * numberIterations) + threadIdx.x * numberIterations;

    if(startEntry < parameterSize) {

        int indexGradient = blockIdx.x;
        int indexParameter = parameterIndices[indexGradient];

        int startParameter = indexParameter * parameterSize + startEntry;
        int startGradient = indexGradient * parameterSize + startEntry;

        for(int i = 0; i < numberIterations; i++) {

            float update = momentum * history[startParameter + i] - scalingFactor * learningRate * gradient[startGradient + i];

            history[startParameter + i] = update;
            parameters[startParameter + i] += update;

        }

    }

}