
#include <hip/hip_runtime.h>
// -1/target probability if target = 1.0, 0.0 otherwise
__global__ void backwardLogisticLossKernel (float *predictions, float *targets, float *result)
{

    int globalId = blockIdx.x * blockDim.x + threadIdx.x;

    result[globalId] = targets[globalId] * -(1.0/predictions[globalId]);

}