
#include <hip/hip_runtime.h>
__device__ double sigmoid (double x)
{

    return 1.0 / (1.0 + exp (-x));

}

extern "C"
__global__ void sigmoidKernel (int length, double *source, double *destination)
{

    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if(threadId < length) {

        destination[threadId] = sigmoid(source[threadId]);

    }

}