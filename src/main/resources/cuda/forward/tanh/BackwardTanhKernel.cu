#include "hip/hip_runtime.h"
#include "symbols/Zero.cuh"

__device__ float backwardTanh (float forward, float chain)
{

    return chain * (1.0 - powf(forward, 2.0));

}

extern "C"
__global__ void backwardTanhKernel (int batchSize, int numberEntriesPerInstance, int numberIterations, float *forward, float *chain, float *destination) {

    int indexInstance = blockIdx.x;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startNextInstanceWithinBatch = startInstanceWithinBatch + numberEntriesPerInstance;

    int firstEntryWithinBatch = startInstanceWithinBatch + blockIdx.y * blockDim.x * numberIterations + threadIdx.x * numberIterations;

    if(firstEntryWithinBatch < startNextInstanceWithinBatch) {

        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextInstanceWithinBatch);

        if(indexInstance < batchSize) {

            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {

                destination[indexEntry] = backwardTanh(forward[indexEntry], chain[indexEntry]);

            }

        }
        else {

            setToZero(destination, firstEntryWithinBatch, lastEntryWithinBatch);

        }

    }

}