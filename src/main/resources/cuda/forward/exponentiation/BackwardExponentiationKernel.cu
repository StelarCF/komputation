#include "hip/hip_runtime.h"
#include "zero/Zero.cuh"

extern "C"
__global__ void backwardExponentiationKernel (
    int batchSize,
    int numberEntriesPerInstance,
    int numberIterations,
    float *forward,
    float *chain,
    float *destination) {

    int indexInstance = blockIdx.x;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startNextInstanceWithinBatch = startInstanceWithinBatch + numberEntriesPerInstance;

    int firstEntryWithinBatch = startInstanceWithinBatch + blockIdx.y * blockDim.x * numberIterations + threadIdx.x * numberIterations;

    if(firstEntryWithinBatch < startNextInstanceWithinBatch) {

        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextInstanceWithinBatch);

        if(indexInstance < batchSize) {

            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {

                destination[indexEntry] = chain[indexEntry] * forward[indexEntry];

            }

        }
        else {

            setToZero(destination, firstEntryWithinBatch, lastEntryWithinBatch);

        }

    }

}