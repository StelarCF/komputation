#include "hip/hip_runtime.h"
#include "zero/Zero.cuh"

extern "C"
__global__ void exponentiationKernel (
    int batchSize,
    int numberEntriesPerInstance,
    int numberIterations,
    float *source,
    float *destination) {

    int indexInstance = blockIdx.x;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startNextInstanceWithinBatch = startInstanceWithinBatch + numberEntriesPerInstance;

    int firstEntryWithinBatch = startInstanceWithinBatch + blockIdx.y * blockDim.x * numberIterations + threadIdx.x * numberIterations;

    if(firstEntryWithinBatch < startNextInstanceWithinBatch) {

        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextInstanceWithinBatch);

        if(indexInstance < batchSize) {

            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {

                destination[indexEntry] = expf(source[indexEntry]);

            }

        }
        else {

            setToZero(destination, firstEntryWithinBatch, lastEntryWithinBatch);

        }

    }

}