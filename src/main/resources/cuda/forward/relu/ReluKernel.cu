#include "hip/hip_runtime.h"
#include "symbols/Zero.cuh"

__device__ float relu (float x)
{

    return fmaxf(x, 0.0);

}

extern "C"
__global__ void reluKernel (
    int batchSize,
    int numberEntriesPerInstance,
    int numberIterations,
    float *source,
    float *destination) {

    int indexInstance = blockIdx.x;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startNextInstanceWithinBatch = startInstanceWithinBatch + numberEntriesPerInstance;

    int firstEntryWithinBatch = startInstanceWithinBatch + blockIdx.y * blockDim.x * numberIterations + threadIdx.x * numberIterations;

    if(firstEntryWithinBatch < startNextInstanceWithinBatch) {

        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextInstanceWithinBatch);

        if(indexInstance < batchSize) {

            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {

                destination[indexEntry] = relu(source[indexEntry]);

            }

        }
        else {

            setToZero(destination, firstEntryWithinBatch, lastEntryWithinBatch);

        }

    }

}