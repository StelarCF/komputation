#include "hip/hip_runtime.h"
#include "zero/Zero.cuh"

__device__ float backwardRelu (float forward, float chain)
{

    if(forward > 0.0) {

        return chain;

    }
    else {

        return 0.0;

    }

}

extern "C"
__global__ void backwardReluKernel (int batchSize, int numberEntriesPerInstance, int numberIterations, float *forward, float *chain, float *destination) {

    // What's the first entry index within the instance that this thread should operate on?
    int startIndexWithinInstance = blockIdx.y * (blockDim.x * numberIterations) + threadIdx.x * numberIterations;

    // Continue if this index is smaller than the dimension of the instance.
    if(startIndexWithinInstance < numberEntriesPerInstance) {

        // What's the first entry index within the batch that this thread should operate on?
        int startIndexWithinBatch = blockIdx.x * numberEntriesPerInstance + startIndexWithinInstance;

        // Is the instance greater than the current batch size?
        if(blockIdx.x >= batchSize) {

            setToZero(destination, startIndexWithinBatch, numberIterations);

        }
        else {

            for(int indexEntry = startIndexWithinBatch; indexEntry < startIndexWithinBatch + numberIterations; indexEntry++) {

                destination[indexEntry] = backwardRelu(forward[indexEntry], chain[indexEntry]);

            }

        }

    }

}