#include "hip/hip_runtime.h"
#include "symbols/Zero.cuh"

__device__ float backwardRelu (float forward, float chain)
{

    if(forward > 0.0) {

        return chain;

    }
    else {

        return 0.0;

    }

}

extern "C"
__global__ void backwardReluKernel (int batchSize, int numberEntriesPerInstance, int numberIterations, float *forward, float *chain, float *destination) {

    int indexInstance = blockIdx.x;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startNextInstanceWithinBatch = startInstanceWithinBatch + numberEntriesPerInstance;

    int firstEntryWithinBatch = startInstanceWithinBatch + blockIdx.y * blockDim.x * numberIterations + threadIdx.x * numberIterations;

    if(firstEntryWithinBatch < startNextInstanceWithinBatch) {

        int lastEntryWithinBatch = min(firstEntryWithinBatch + numberIterations, startNextInstanceWithinBatch);

        if(indexInstance < batchSize) {

            for(int indexEntry = firstEntryWithinBatch; indexEntry < lastEntryWithinBatch; indexEntry++) {

                destination[indexEntry] = backwardRelu(forward[indexEntry], chain[indexEntry]);

            }

        }
        else {

            setToZero(destination, firstEntryWithinBatch, lastEntryWithinBatch);

        }

    }

}