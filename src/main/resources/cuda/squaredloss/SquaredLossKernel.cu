#include "hip/hip_runtime.h"
#include "reduction/Reduction.cuh"

// This assumes that the number of threads is equal to the number of predictions/targets.
// First, the squared differences between predictions and targets are stored in shared memory.
// In the second step, the squared differences are summed up using a parallel reduction.
// Finally, the sum is multiplied by 1/2.
template <int blockSize>
__global__ void squaredLossKernel (int numberEntries, float *predictions, float *targets, float *result)
{

    int threadId = threadIdx.x;

    extern __shared__ float sharedData[];

    if(threadId < numberEntries) {

        sharedData[threadId] = powf(predictions[threadId] - targets[threadId], 2.0);

    }

    __syncthreads();

    reduce<blockSize>(threadId, sharedData, 0);

    if(threadId == 0) {

        result[0] = 0.5 * sharedData[0];

    }

}