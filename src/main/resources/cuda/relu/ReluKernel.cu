#include "hip/hip_runtime.h"
#include "zero/Zero.cuh"

__device__ float relu (float x)
{

    return fmaxf(x, 0.0);

}

extern "C"
__global__ void reluKernel (int batchSize, int numberEntriesPerInstance, int numberIterations, float *source, float *destination) {

    // What's the first entry index within the instance that this thread should operate on?
    int startIndexWithinInstance = blockIdx.y * (blockDim.x * numberIterations) + threadIdx.x * numberIterations;

    // Continue if this index is smaller than the dimension of the instance.
    if(startIndexWithinInstance < numberEntriesPerInstance) {

        // What's the first entry index within the batch that this thread should operate on?
        int startIndexWithinBatch = blockIdx.x * numberEntriesPerInstance + startIndexWithinInstance;

        // Is the instance greater than the current batch size?
        if(blockIdx.x >= batchSize) {

            setToZero(destination, startIndexWithinBatch, numberIterations);

        }
        else {

            for(int indexEntry = startIndexWithinBatch; indexEntry < startIndexWithinBatch + numberIterations; indexEntry++) {

                destination[indexEntry] = relu(source[indexEntry]);

            }

        }

    }

}