
#include <hip/hip_runtime.h>
__device__ float relu (float x)
{

    return fmaxf(x, 0.0);

}

extern "C"
__global__ void reluKernel (int batchSize, int numberEntriesPerInstance, float *source, float *destination)
{

    int indexInstance = blockIdx.x;
    int startInstance = indexInstance * numberEntriesPerInstance;
    int indexEntryInInstance = blockIdx.y * blockDim.y + threadIdx.x;
    int indexEntryInBatch = startInstance + indexEntryInInstance;

    if(indexEntryInInstance < numberEntriesPerInstance) {

        if(indexInstance < batchSize) {

            destination[indexEntryInBatch] = relu(source[indexEntryInBatch]);

        }
        else {

            destination[indexEntryInBatch] = 0.0;

        }

    }

}